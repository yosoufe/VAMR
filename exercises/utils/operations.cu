#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/iterator/zip_iterator.h>

#include "operations.cuh"
#include "operations.hpp"
#include "utils.cuh"
#include <Eigen/Dense>

cuda::CuMatrixD cuda::sobel_x_kernel()
{
    Eigen::MatrixXd kernel = ::sobel_x_kernel();
    return cuda::eigen_to_cuda(kernel);
}

cuda::CuMatrixD cuda::sobel_y_kernel()
{
    Eigen::MatrixXd kernel = ::sobel_y_kernel();
    return cuda::eigen_to_cuda(kernel);
}

cuda::CuMatrixD cuda::ones(int rows, int cols)
{
    Eigen::MatrixXd m = Eigen::MatrixXd::Ones(rows, cols);
    return cuda::eigen_to_cuda(m);
}

void setTensorDesc(hipdnnTensorDescriptor_t &tensorDesc,
                   const hipdnnTensorFormat_t &tensorFormat,
                   const hipdnnDataType_t &dataType,
                   int n,
                   int c,
                   int h,
                   int w)
{
#define ND_TENSOR_DESCRIPTOR
#if SIMPLE_TENSOR_DESCRIPTOR
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(tensorDesc,
                                          tensorFormat,
                                          dataType,
                                          n, c,
                                          h,
                                          w));
#elif defined(ND_TENSOR_DESCRIPTOR)
    const int nDims = 4;
    int dimA[nDims] = {n, c, h, w};
    int strideA[nDims] = {c * h * w, h * w, w, 1};
    CUDNN_CALL(hipdnnSetTensorNdDescriptor(tensorDesc,
                                          dataType,
                                          4,
                                          dimA,
                                          strideA));
#else
    CUDNN_CALL(hipdnnSetTensor4dDescriptorEx(tensorDesc,
                                            dataType,
                                            n, c,
                                            h, w,
                                            c * h * w, h * w, w, 1));
#endif
}

cuda::CuMatrixD cuda::correlation(const cuda::CuMatrixD &input, const cuda::CuMatrixD &kernel)
{
    /**
     * in CUDNN it seems it is
     * height <-> number of columns
     * weight <-> number of rows.
     */
    hipdnnHandle_t cudnn;
    CUDNN_CALL(hipdnnCreate(&cudnn));
    CUDNN_CALL(cudnnCnnInferVersionCheck());

    // input
    const int in_n = 1;
    const int in_c = 1;
    const int in_h = input.cols();
    const int in_w = input.rows();

    hipdnnTensorDescriptor_t in_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
    // CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    //     in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE,
    //     in_n, in_c, in_h, in_w));

    setTensorDesc(in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE,
                  in_n, in_c, in_h, in_w);

    double *in_data = input.data();

    // filter
    const int filt_k = 1;
    const int filt_c = 1;
    const int filt_h = kernel.cols();
    const int filt_w = kernel.rows();

    hipdnnFilterDescriptor_t filt_desc;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(
        filt_desc, HIPDNN_DATA_DOUBLE, HIPDNN_TENSOR_NCHW,
        filt_k, filt_c, filt_h, filt_w));

    double *filt_data = kernel.data();

    // convolution
    hipdnnConvolutionDescriptor_t conv_desc;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));

    const int convDims = 2;
    int padA[convDims] = {kernel.cols() / 2, kernel.rows() / 2};
    int filterStrideA[convDims] = {1, 1};
    int upscaleA[convDims] = {1, 1};
    CUDNN_CALL(hipdnnSetConvolutionNdDescriptor(
        conv_desc, convDims, padA, filterStrideA,
        upscaleA, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_DOUBLE));

    // output
    int out_n;
    int out_c;
    int out_h;
    int out_w;

    const int tensorDims = 4;
    int tensorOuputDimA[tensorDims];
    CUDNN_CALL(cudnnGetConvolutionNdForwardOutputDim(
        conv_desc, in_desc, filt_desc,
        tensorDims, tensorOuputDimA));

    out_n = tensorOuputDimA[0];
    out_c = tensorOuputDimA[1];
    out_h = tensorOuputDimA[2];
    out_w = tensorOuputDimA[3];

    // std::cout << " out_n " << out_n << " out_c " << out_c;
    // std::cout << " out_h " << out_h << " out_w " << out_w << std::endl;

    hipdnnTensorDescriptor_t out_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));

    setTensorDesc(out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE,
                  out_n, out_c, out_h, out_w);

    double *out_data;
    CSC(hipMalloc(
        &out_data, out_n * out_c * out_h * out_w * sizeof(double)));

    // algorithm
    hipdnnConvolutionFwdAlgo_t algo;
    int requestedAlgoCount = HIPDNN_CONVOLUTION_FWD_ALGO_COUNT;
    int returnedAlgoCount = -1;
    hipdnnConvolutionFwdAlgoPerf_t results[2 * HIPDNN_CONVOLUTION_FWD_ALGO_COUNT];
    CUDNN_CALL(hipdnnFindConvolutionForwardAlgorithm(
        cudnn,
        in_desc, filt_desc, conv_desc, out_desc,
        requestedAlgoCount, &returnedAlgoCount, results));
    algo = results[0].algo;

    // workspace
    size_t ws_size;
    CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
        cudnn, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));

    double *ws_data;
    if (ws_size != 0)
        CSC(hipMalloc(&ws_data, ws_size));

    // perform
    double alpha = 1.0;
    double beta = 0;

    CUDNN_CALL(hipdnnConvolutionForward(
        cudnn,
        &alpha, in_desc, in_data, filt_desc, filt_data,
        conv_desc, algo, ws_data, ws_size,
        &beta, out_desc, out_data));

    // zeros out the elements that are calculated by the padding
    auto out = cuda::CuMatrixD(out_data, out_w, out_h);
    int s_row = kernel.rows() / 2;
    int s_col = kernel.cols() / 2;
    int l_row = input.rows() - 2 * s_row;
    int l_col = input.cols() - 2 * s_col;

    zero_borders(out, s_row, s_col, l_row, l_col);

    // finalizing
    if (ws_size != 0)
        CSC(hipFree(ws_data));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(out_desc));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filt_desc));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(in_desc));
    CUDNN_CALL(hipdnnDestroy(cudnn));

    return out;
}

template <typename functor>
void _unary_operator(const cuda::CuMatrixD &input, cuda::CuMatrixD &output, functor unary_f)
{
    auto d_vec_start = cuda::thrust_ptr_begin(input);
    auto d_vec_end = cuda::thrust_ptr_end(input);
    auto d_output_start = cuda::thrust_ptr_begin(output);
    thrust::transform(thrust::cuda::par, d_vec_start, d_vec_end, d_output_start, unary_f);
}

template <typename T>
struct power
{
    T p;
    power(T p) : p(p){};
    __host__ __device__ T operator()(const T &x) const
    {
        return pow(x, p);
    }
};

cuda::CuMatrixD cuda::pow(const cuda::CuMatrixD &input, double pow)
{
    cuda::CuMatrixD res(input.rows(), input.cols());
    _unary_operator(input, res, power<double>(pow));
    return res;
}

cuda::CuMatrixD cuda::pow(cuda::CuMatrixD &&input, double pow)
{
    _unary_operator(input, input, power<double>(pow));
    return input;
}

double cuda::norm(const cuda::CuMatrixD &input)
{
    auto squared = cuda::pow(input, 2);
    auto sum_of_squared = thrust::reduce(
        thrust::cuda::par,
        cuda::thrust_ptr_begin(squared), 
        cuda::thrust_ptr_end(squared));
    return std::sqrt(sum_of_squared);
}

template <typename functor>
void _binary_operator(const cuda::CuMatrixD &i1, const cuda::CuMatrixD &i2, cuda::CuMatrixD &output, functor binary_f)
{
    auto s1 = cuda::thrust_ptr_begin(i1);
    auto e1 = cuda::thrust_ptr_end(i1);
    auto s2 = cuda::thrust_ptr_begin(i2);
    auto output_ptr = cuda::thrust_ptr_begin(output);
    thrust::transform(thrust::cuda::par, s1, e1, s2, output_ptr, binary_f);
}

template <typename T>
struct multiply_functor
    : public thrust::binary_function<T, T, T>
{
    __host__ __device__ T operator()(T x, T y)
    {
        return x * y;
    }
};

cuda::CuMatrixD cuda::operator*(const cuda::CuMatrixD &i1, const cuda::CuMatrixD &i2)
{
    cuda::CuMatrixD out(i1.rows(), i1.cols());
    _binary_operator(i1, i2, out, multiply_functor<double>());
    return out;
}

cuda::CuMatrixD cuda::operator*(cuda::CuMatrixD &&i1, const cuda::CuMatrixD &i2)
{
    _binary_operator(i1, i2, i1, multiply_functor<double>());
    return i1;
}

cuda::CuMatrixD cuda::operator*(const cuda::CuMatrixD &i1, cuda::CuMatrixD &&i2)
{
    _binary_operator(i1, i2, i2, multiply_functor<double>());
    return i2;
}

cuda::CuMatrixD cuda::operator*(cuda::CuMatrixD &&i1, cuda::CuMatrixD &&i2)
{
    _binary_operator(i1, i2, i1, multiply_functor<double>());
    return i1;
}

template <typename T>
struct plus_functor
    : public thrust::binary_function<T, T, T>
{
    __host__ __device__ T operator()(T x, T y)
    {
        return x + y;
    }
};

cuda::CuMatrixD cuda::operator+(const cuda::CuMatrixD &i1, const cuda::CuMatrixD &i2)
{
    cuda::CuMatrixD out(i1.rows(), i1.cols());
    _binary_operator(i1, i2, out, plus_functor<double>());
    return out;
}

cuda::CuMatrixD cuda::operator+(cuda::CuMatrixD &&i1, const cuda::CuMatrixD &i2)
{
    _binary_operator(i1, i2, i1, plus_functor<double>());
    return i1;
}

cuda::CuMatrixD cuda::operator+(const cuda::CuMatrixD &i1, cuda::CuMatrixD &&i2)
{
    _binary_operator(i1, i2, i2, plus_functor<double>());
    return i2;
}

cuda::CuMatrixD cuda::operator+(cuda::CuMatrixD &&i1, cuda::CuMatrixD &&i2)
{
    _binary_operator(i1, i2, i1, plus_functor<double>());
    return i1;
}

template <typename T>
struct minus_functor
    : public thrust::binary_function<T, T, T>
{
    __host__ __device__ T operator()(T x, T y)
    {
        return x - y;
    }
};

cuda::CuMatrixD cuda::operator-(const cuda::CuMatrixD &i1, const cuda::CuMatrixD &i2)
{
    cuda::CuMatrixD out(i1.rows(), i1.cols());
    _binary_operator(i1, i2, out, minus_functor<double>());
    return out;
}

cuda::CuMatrixD cuda::operator-(cuda::CuMatrixD &&i1, const cuda::CuMatrixD &i2)
{
    _binary_operator(i1, i2, i1, minus_functor<double>());
    return i1;
}

cuda::CuMatrixD cuda::operator-(const cuda::CuMatrixD &i1, cuda::CuMatrixD &&i2)
{
    _binary_operator(i1, i2, i2, minus_functor<double>());
    return i2;
}

cuda::CuMatrixD cuda::operator-(cuda::CuMatrixD &&i1, cuda::CuMatrixD &&i2)
{
    _binary_operator(i1, i2, i1, minus_functor<double>());
    return i1;
}

template <typename T>
struct multiply_by_constant
{
    T cst;
    multiply_by_constant(T cst) : cst(cst){};
    __host__ __device__ T operator()(const T &x) const
    {
        return x * cst;
    }
};

cuda::CuMatrixD cuda::operator*(const cuda::CuMatrixD &mat, double constant)
{
    cuda::CuMatrixD out(mat.rows(), mat.cols());
    _unary_operator(mat, out, multiply_by_constant<double>(constant));
    return out;
}

cuda::CuMatrixD cuda::operator*(double constant, const cuda::CuMatrixD &mat)
{
    return cuda::operator*(mat, constant);
}

cuda::CuMatrixD cuda::operator*(cuda::CuMatrixD &&mat, double constant)
{
    _unary_operator(mat, mat, multiply_by_constant<double>(constant));
    return mat;
}

cuda::CuMatrixD cuda::operator*(double constant, cuda::CuMatrixD &&mat)
{
    _unary_operator(mat, mat, multiply_by_constant<double>(constant));
    return mat;
}

template <typename T>
struct thrshold_lower_functor
{
    T threshold;
    T substitute;
    thrshold_lower_functor(T th, T sub) : threshold(th), substitute(sub){};
    __host__ __device__ T operator()(const T &x) const
    {
        if (x < threshold)
            return substitute;
        return x;
    }
};

cuda::CuMatrixD cuda::threshold_lower(const cuda::CuMatrixD &input, double threshold, double substitute)
{
    cuda::CuMatrixD out(input.rows(), input.cols());
    _unary_operator(input, out, thrshold_lower_functor<double>(threshold, substitute));
    return out;
}

cuda::CuMatrixD cuda::threshold_lower(cuda::CuMatrixD &&input, double threshold, double substitute)
{
    _unary_operator(input, input, thrshold_lower_functor<double>(threshold, substitute));
    return input;
}

template <typename T>
struct ZeroBorderOperator
{
    int n_rows, n_cols;
    int s_row, s_col, l_row, l_col;
    ZeroBorderOperator(int n_rows, int n_cols,
                       int s_row, int s_col,
                       int l_row, int l_col) : n_rows(n_rows),
                                               n_cols(n_cols),
                                               s_row(s_row),
                                               s_col(s_col),
                                               l_row(l_row),
                                               l_col(l_col)
    {
    }

    template <typename Tuple>
    __host__ __device__ T operator()(Tuple t)
    {
        int index = thrust::get<1>(t);
        T value = thrust::get<0>(t);
        auto idx = get_2d_index_colwise(index, n_rows);
        auto row = thrust::get<0>(idx);
        auto col = thrust::get<1>(idx);

        if (col < s_col || col >= s_col + l_col || row < s_row || row >= s_row + l_row)
            return T(0.0);
        else
            return value;
    }
};

thrust::device_ptr<int> cuda::create_indices(const cuda::CuMatrixD &input)
{
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(input.n_elements());
    thrust::sequence(thrust::device, d_output, d_output + input.n_elements());
    return d_output;
}

void cuda::zero_borders(cuda::CuMatrixD &input, int s_row, int s_col, int l_row, int l_col)
{
    auto d_indices_start = cuda::create_indices(input);
    auto &output = input;
    thrust::device_ptr<double> d_vec_start = thrust::device_pointer_cast(input.data());
    thrust::device_ptr<double> d_vec_end = d_vec_start + input.cols() * input.rows();
    thrust::device_ptr<double> d_output_start = thrust::device_pointer_cast(output.data());
    ZeroBorderOperator<double> ops(input.rows(), input.cols(), s_row, s_col, l_row, l_col);
    thrust::transform(thrust::cuda::par,
                      thrust::make_zip_iterator(thrust::make_tuple(d_vec_start, d_indices_start)),
                      thrust::make_zip_iterator(thrust::make_tuple(d_vec_end, d_indices_start + input.n_elements())),
                      d_output_start, ops);
}