#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/transform.h>

#include "operations.cuh"
#include "operations.hpp"
#include "utils.cuh"
#include <Eigen/Dense>

cuda::CuMatrixD cuda::sobel_x_kernel()
{
    Eigen::MatrixXd kernel = ::sobel_x_kernel();
    return cuda::eigen_to_cuda(kernel);
}

cuda::CuMatrixD cuda::sobel_y_kernel()
{
    Eigen::MatrixXd kernel = ::sobel_y_kernel();
    return cuda::eigen_to_cuda(kernel);
}

cuda::CuMatrixD cuda::ones(int rows, int cols)
{
    Eigen::MatrixXd m = Eigen::MatrixXd::Ones(rows, cols);
    return cuda::eigen_to_cuda(m);
}

cuda::CuMatrixD cuda::correlation(const cuda::CuMatrixD &input, const cuda::CuMatrixD &kernel)
{
    hipdnnHandle_t cudnn;
    CUDNN_CALL(hipdnnCreate(&cudnn));
    CUDNN_CALL(cudnnCnnInferVersionCheck());

    // input
    const int in_n = 1;
    const int in_c = 1;
    const int in_h = input.n_rows;
    const int in_w = input.n_cols;

    hipdnnTensorDescriptor_t in_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
        in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE,
        in_n, in_c, in_h, in_w));

    double *in_data = input.d_data.get();

    // filter
    const int filt_k = 1;
    const int filt_c = 1;
    const int filt_h = kernel.n_rows;
    const int filt_w = kernel.n_cols;

    hipdnnFilterDescriptor_t filt_desc;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(
        filt_desc, HIPDNN_DATA_DOUBLE, HIPDNN_TENSOR_NCHW,
        filt_k, filt_c, filt_h, filt_w));

    double *filt_data = kernel.d_data.get();

    // convolution
    const int pad_h = kernel.n_cols / 2;
    const int pad_w = kernel.n_rows / 2;
    const int str_h = 1;
    const int str_w = 1;
    const int dil_h = 1;
    const int dil_w = 1;

    hipdnnConvolutionDescriptor_t conv_desc;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
        conv_desc,
        pad_h, pad_w, str_h, str_w, dil_h, dil_w,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_DOUBLE));

    // output
    int out_n;
    int out_c;
    int out_h;
    int out_w;

    CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(
        conv_desc, in_desc, filt_desc,
        &out_n, &out_c, &out_h, &out_w));

    hipdnnTensorDescriptor_t out_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
        out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE,
        out_n, out_c, out_h, out_w));

    double *out_data;
    CSC(hipMalloc(
        &out_data, out_n * out_c * out_h * out_w * sizeof(double)));

    // algorithm
    hipdnnConvolutionFwdAlgoPerf_t algo;
    int perf_count;
    CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(
        cudnn,
        in_desc, filt_desc, conv_desc, out_desc,
        1, &perf_count, &algo));

    // workspace
    size_t ws_size;
    CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
        cudnn, in_desc, filt_desc, conv_desc, out_desc, algo.algo, &ws_size));

    double *ws_data;
    CSC(hipMalloc(&ws_data, ws_size));

    // perform
    double alpha = 1.0;
    double beta = 0;

    CUDNN_CALL(hipdnnConvolutionForward(
        cudnn,
        &alpha, in_desc, in_data, filt_desc, filt_data,
        conv_desc, algo.algo, ws_data, ws_size,
        &beta, out_desc, out_data));

    auto out = cuda::CuMatrixD(out_data, out_w, out_h);

    // finalizing
    CSC(hipFree(ws_data));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(out_desc));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filt_desc));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(in_desc));
    CUDNN_CALL(hipdnnDestroy(cudnn));

    return out;
}

template <typename functor>
void _unary_operator(const cuda::CuMatrixD &input,cuda::CuMatrixD &output, functor unary_f)
{
    thrust::device_ptr<double> d_vec_start = thrust::device_pointer_cast(input.d_data.get());
    thrust::device_ptr<double> d_vec_end = d_vec_start + input.n_cols * input.n_rows;
    thrust::device_ptr<double> d_output_start = thrust::device_pointer_cast(output.d_data.get());
    thrust::transform(thrust::cuda::par, d_vec_start, d_vec_end, d_output_start, unary_f);
}


template <typename T>
struct power
{
    T p;
    power(T p) : p(p){};
    __host__ __device__ T operator()(const T &x) const
    {
        return pow(x, p);
    }
};

cuda::CuMatrixD cuda::pow(const cuda::CuMatrixD &input, double pow)
{
    cuda::CuMatrixD res(input.n_cols, input.n_rows);
    _unary_operator(input, res, power<double>(pow));
    return res;
}


cuda::CuMatrixD cuda::pow(cuda::CuMatrixD &&input, double pow)
{
    _unary_operator(input, input, power<double>(pow));
    return input;
}

template <typename functor>
void _binary_operator(const cuda::CuMatrixD &i1, const cuda::CuMatrixD &i2, cuda::CuMatrixD &output, functor binary_f)
{
    thrust::device_ptr<double> s1 = thrust::device_pointer_cast(i1.d_data.get());
    thrust::device_ptr<double> e1 = s1 + i1.n_cols * i1.n_rows;
    thrust::device_ptr<double> s2 = thrust::device_pointer_cast(i2.d_data.get());
    thrust::device_ptr<double> output_ptr = thrust::device_pointer_cast(output.d_data.get());
    thrust::transform(thrust::cuda::par, s1, e1, s2, output_ptr, binary_f);
}

template <typename T>
struct multiply_functor
    : public thrust::binary_function<T, T, T>
{
    __host__ __device__ T operator()(T x, T y)
    {
        return x * y;
    }
};

cuda::CuMatrixD cuda::operator*(const cuda::CuMatrixD &i1, const cuda::CuMatrixD &i2)
{
    cuda::CuMatrixD out(i1.n_cols, i1.n_rows);
    _binary_operator(i1, i2, out, multiply_functor<double>());
    return out;
}

cuda::CuMatrixD cuda::operator*(cuda::CuMatrixD &&i1, const cuda::CuMatrixD &i2)
{
    _binary_operator(i1, i2, i1, multiply_functor<double>());
    return i1;
}

cuda::CuMatrixD cuda::operator*(const cuda::CuMatrixD &i1, cuda::CuMatrixD &&i2)
{
    _binary_operator(i1, i2, i2, multiply_functor<double>());
    return i2;
}

cuda::CuMatrixD cuda::operator*(cuda::CuMatrixD &&i1, cuda::CuMatrixD &&i2)
{
    _binary_operator(i1, i2, i1, multiply_functor<double>());
    return i1;
}

template <typename T>
struct plus_functor
    : public thrust::binary_function<T, T, T>
{
    __host__ __device__ T operator()(T x, T y)
    {
        return x + y;
    }
};

cuda::CuMatrixD cuda::operator+(const cuda::CuMatrixD &i1, const cuda::CuMatrixD &i2)
{
    cuda::CuMatrixD out(i1.n_cols, i1.n_rows);
    _binary_operator(i1, i2, out, plus_functor<double>());
    return out;
}

cuda::CuMatrixD cuda::operator+(cuda::CuMatrixD &&i1, const cuda::CuMatrixD &i2)
{
    _binary_operator(i1, i2, i1, plus_functor<double>());
    return i1;
}

cuda::CuMatrixD cuda::operator+(const cuda::CuMatrixD &i1, cuda::CuMatrixD &&i2)
{
    _binary_operator(i1, i2, i2, plus_functor<double>());
    return i2;
}

cuda::CuMatrixD cuda::operator+(cuda::CuMatrixD &&i1, cuda::CuMatrixD &&i2)
{
    _binary_operator(i1, i2, i1, plus_functor<double>());
    return i1;
}

template <typename T>
struct minus_functor
    : public thrust::binary_function<T, T, T>
{
    __host__ __device__ T operator()(T x, T y)
    {
        return x - y;
    }
};

cuda::CuMatrixD cuda::operator-(const cuda::CuMatrixD &i1, const cuda::CuMatrixD &i2)
{
    cuda::CuMatrixD out(i1.n_cols, i1.n_rows);
    _binary_operator(i1, i2, out, minus_functor<double>());
    return out;
}

cuda::CuMatrixD cuda::operator-(cuda::CuMatrixD &&i1, const cuda::CuMatrixD &i2)
{
    _binary_operator(i1, i2, i1, minus_functor<double>());
    return i1;
}

cuda::CuMatrixD cuda::operator-(const cuda::CuMatrixD &i1, cuda::CuMatrixD &&i2)
{
    _binary_operator(i1, i2, i2, minus_functor<double>());
    return i2;
}

cuda::CuMatrixD cuda::operator-(cuda::CuMatrixD &&i1, cuda::CuMatrixD &&i2)
{
    _binary_operator(i1, i2, i1, minus_functor<double>());
    return i1;
}

template <typename T>
struct multiply_by_constant
{
    T cst;
    multiply_by_constant(T cst) : cst(cst){};
    __host__ __device__ T operator()(const T &x) const
    {
        return x * cst;
    }
};


cuda::CuMatrixD cuda::operator*(const cuda::CuMatrixD &mat, double constant)
{
    cuda::CuMatrixD out(mat.n_cols, mat.n_rows);
    _unary_operator(mat, out, multiply_by_constant<double>(constant));
    return out;
}

cuda::CuMatrixD cuda::operator*(double constant, const cuda::CuMatrixD &mat)
{
    return cuda::operator*(mat, constant);
}

cuda::CuMatrixD cuda::operator*(cuda::CuMatrixD &&mat, double constant)
{
    _unary_operator(mat, mat, multiply_by_constant<double>(constant));
    return mat;
}

cuda::CuMatrixD cuda::operator*(double constant, cuda::CuMatrixD &&mat)
{
    _unary_operator(mat, mat, multiply_by_constant<double>(constant));
    return mat;
}

template <typename T>
struct thrshold_lower_functor
{
    T threshold;
    T substitute;
    thrshold_lower_functor(T th, T sub) : threshold(th), substitute(sub){};
    __host__ __device__ T operator()(const T &x) const
    {
        if (x < threshold)
            return substitute;
        return x;
    }
};


cuda::CuMatrixD cuda::threshold_lower(const cuda::CuMatrixD &input, double threshold, double substitute)
{
    cuda::CuMatrixD out(input.n_cols, input.n_rows);
    _unary_operator(input, out, thrshold_lower_functor<double>(threshold, substitute));
    return out;
}

cuda::CuMatrixD cuda::threshold_lower(cuda::CuMatrixD &&input, double threshold, double substitute)
{
    _unary_operator(input, input, thrshold_lower_functor<double>(threshold, substitute));
    return input;
}