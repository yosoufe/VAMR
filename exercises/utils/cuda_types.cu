#include "hip/hip_runtime.h"
#include "cuda_types.cuh"
#include "cuda_types.hpp"
#include "utils.cuh"
#include "utils.hpp"

template <typename T>
void cuda::CuMatrixDeleter<T>::operator()(T *p) const
{
    if (p != nullptr)
    {
        // FIXME: this should not throw exception.
        // or exit. CSC might exit.
        // std::cout << "freeing gpu memory" << std::endl;
        CSC(hipFree(p));
        p = nullptr;
    }
}

template <typename T>
cuda::CuMatrix<T>::CuMatrix() : cuda::CuMatrix<T>(nullptr, 0, 0)
{
}

template <typename T>
cuda::CuMatrix<T>::CuMatrix(T *ptr, int n_cols, int n_rows) : d_data(
                                                                  std::shared_ptr<T>(ptr,
                                                                                     cuda::CuMatrixDeleter<T>())),
                                                              n_cols(n_cols),
                                                              n_rows(n_rows)
{
}

template <typename T>
cuda::CuMatrix<T>::CuMatrix(int cols, int rows) : n_cols(cols), n_rows(rows)
{
    int number_of_bytes = sizeof(T) * cols * rows;
    T *ptr;
    CSC(hipMalloc(&ptr, number_of_bytes));
    d_data = std::shared_ptr<T>(ptr,
                                cuda::CuMatrixDeleter<T>());
}

template <typename T>
cuda::CuMatrix<T> cuda::CuMatrix<T>::clone() const
{
    cuda::CuMatrix<T> output(n_cols, n_rows);
    CSC(hipMemcpy(output.d_data.get(), d_data.get(), sizeof(T) * n_elements(), hipMemcpyDeviceToDevice));
    return output;
}

template <typename T>
__global__ void
copy_block_kernel(T *src, T *dst,
                  int src_n_rows,
                  int src_row_offset, int src_col_offset,
                  int dst_n_rows, int dst_n_cols)
{
    int dst_row = threadIdx.x + blockDim.x * blockIdx.x;
    int dst_col = threadIdx.y + blockDim.y * blockIdx.y;

    if (dst_row >= dst_n_rows || dst_col >= dst_n_cols)
        return;

    int src_row = dst_row + src_row_offset;
    int src_col = dst_col + src_col_offset;
    dst[get_index_colwise(dst_row, dst_col, dst_n_rows)] =
        src[get_index_colwise(src_row, src_col, src_n_rows)];
}

template <typename T>
cuda::CuMatrix<T> cuda::CuMatrix<T>::block(int row, int col, int height, int width) const
{
    cuda::CuMatrix<T> output(width, height);

    // super slow for large matrices.
    auto using_stream_impl = [&]()
    {
        hipStream_t streams[width];
        int counter = 0;
        for (int current_col = col; current_col < col + width; ++current_col, ++counter)
        {
            T *src = d_data.get() +
                     get_index_colwise(row, current_col, n_rows);
            T *dst = output.d_data.get() +
                     get_index_colwise(0, counter, height);
            CSC(hipStreamCreate(&streams[counter]));
            CSC(hipMemcpyAsync(dst, src, height * sizeof(T), hipMemcpyDeviceToDevice, streams[counter]));
            hipDeviceSynchronize();
        }
    };

    auto using_kernel_impl = [&]()
    {
        dim3 block_dim;
        block_dim.x = min(32, height);
        block_dim.y = min(32, width);

        dim3 grid_dim;
        grid_dim.x = height / block_dim.x + 1;
        grid_dim.y = width / block_dim.y + 1;

        copy_block_kernel<T><<<grid_dim, block_dim>>>(
            d_data.get(), output.d_data.get(),
            n_rows,
            row, col,
            height, width);
    };

    using_stream_impl();
    // using_kernel_impl();

    return output;
}

template struct cuda::CuMatrixDeleter<double>;
template struct cuda::CuMatrixDeleter<float>;
template struct cuda::CuMatrix<double>;
template struct cuda::CuMatrix<float>;

template <typename T>
cuda::CuMatrix<T> cuda::eigen_to_cuda(const MatrixT<T> &eigen)
{
    int number_of_bytes = sizeof(T) * eigen.cols() * eigen.rows();
    T *output_ptr;
    CSC(hipMalloc(&output_ptr, number_of_bytes));
    CSC(hipMemcpy(output_ptr, eigen.data(), number_of_bytes, hipMemcpyHostToDevice));
    // print_cuda_eigen<T><<<1, 1>>>(cuda_eigen.d_data.get(), eigen.cols(), eigen.rows());
    hipDeviceSynchronize();
    return cuda::CuMatrix<T>(output_ptr, eigen.cols(), eigen.rows());
}

// instantiate template function above
template cuda::CuMatrix<double> cuda::eigen_to_cuda<double>(const MatrixT<double> &);
template cuda::CuMatrix<float> cuda::eigen_to_cuda<float>(const MatrixT<float> &);

template <typename T>
MatrixT<T> cuda::cuda_to_eigen(const cuda::CuMatrix<T> &cuda_eigen)
{
    size_t s = cuda_eigen.n_cols * cuda_eigen.n_rows;
    T *h_data = new T[s];
    int number_of_bytes = sizeof(T) * s;
    CSC(hipMemcpy(h_data, cuda_eigen.d_data.get(), number_of_bytes, hipMemcpyDeviceToHost));
    MatrixT<T> res;
    res = MatrixT<T>::Map(h_data, cuda_eigen.n_rows, cuda_eigen.n_cols);
    return res;
}

// instantiate template function above
template MatrixT<double> cuda::cuda_to_eigen(const cuda::CuMatrix<double> &cuda_eigen);
template MatrixT<float> cuda::cuda_to_eigen(const cuda::CuMatrix<float> &cuda_eigen);

bool are_matrices_close(const cuda::CuMatrixD &first, const Eigen::MatrixXd &second)
{
    auto host_matrix = cuda::cuda_to_eigen(first);
    return are_matrices_close(host_matrix, second);
}